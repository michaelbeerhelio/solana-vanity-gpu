
#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    printf("Found %d CUDA devices\n", deviceCount);
    
    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device %d: %s\n", i, prop.name);
        printf("  Compute capability: %d.%d\n", prop.major, prop.minor);
        printf("  Max threads per block: %d\n", prop.maxThreadsPerBlock);
        printf("  Max threads dimensions: (%d, %d, %d)\n", 
               prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    }
    return 0;
}
