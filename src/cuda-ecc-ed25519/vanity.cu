#include "hip/hip_runtime.h"
#include <vector>
#include <random>
#include <chrono>

#include <iostream>
#include <ctime>

#include <assert.h>
#include <inttypes.h>
#include <pthread.h>
#include <stdio.h>

#include "hiprand/hiprand_kernel.h"
#include "ed25519.h"
#include "fixedint.h"
#include "gpu_common.h"
#include "gpu_ctx.h"

#include "keypair.cu"
#include "sc.cu"
#include "fe.cu"
#include "ge.cu"
#include "sha512.cu"
#include "../config.h"
#define MAX_NUM_GPUS 8

/* -- Types ----------------------------------------------------------------- */

typedef struct {
	// CUDA Random States.
	hiprandState*    states[MAX_NUM_GPUS];
	int            gpuCount;
} config;

/* -- Prototypes, Because C++ ----------------------------------------------- */

void            vanity_setup(config& vanity);
void            vanity_run(config& vanity);
void __global__ vanity_init(unsigned long long int seed, hiprandState* state);
void __global__ vanity_scan(hiprandState* state, int* keys_found, int* gpu, int* execution_count);
bool __device__ b58enc(char* b58, size_t* b58sz, uint8_t* data, size_t binsz);

/* -- Entry Point ----------------------------------------------------------- */

int main(int argc, char const* argv[]) {
	ed25519_set_verbose(true);

	config vanity;
	vanity_setup(vanity);
	vanity_run(vanity);
}

// SMITH
std::string getTimeStr(){
    std::time_t now = std::chrono::system_clock::to_time_t(std::chrono::system_clock::now());
    std::string s(30, '\0');
    std::strftime(&s[0], s.size(), "%Y-%m-%d %H:%M:%S", std::localtime(&now));
    return s;
}

// SMITH - safe? who knows
unsigned long long int makeSeed() {
    unsigned long long int seed = 0;
    char *pseed = (char *)&seed;

    std::random_device rd;

    for(unsigned int b=0; b<sizeof(seed); b++) {
      auto r = rd();
      char *entropy = (char *)&r;
      pseed[b] = entropy[0];
    }

    return seed;
}

/* -- Vanity Step Functions ------------------------------------------------- */

void vanity_setup(config &vanity) {
	printf("GPU: Initializing Memory\n");
	
	// Force CUDA to initialize all devices
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	vanity.gpuCount = deviceCount;
	printf("Detected %d GPUs\n", deviceCount);
	
	// Initialize hiprand states for each GPU
	for (int i = 0; i < vanity.gpuCount; ++i) {
		hipSetDevice(i);
		
		// Get device properties
		hipDeviceProp_t device;
		hipGetDeviceProperties(&device, i);
		
		// Calculate number of threads per block and blocks per grid
		int blockSize = 0, minGridSize = 0;
		hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, vanity_scan, 0, 0);
		
		// Allocate and initialize random states for this GPU
		hiprandState* states;
		int numThreads = blockSize * minGridSize;
		hipMalloc(&states, numThreads * sizeof(hiprandState));
		
		// Initialize random states
		unsigned long long int seed = makeSeed();
		printf("Initialising GPU %d from entropy: %llu\n", i, seed);
		vanity_init<<<minGridSize, blockSize>>>(seed, states);
		
		// Store the states in our config
		vanity.states[i] = states;
		
		printf("GPU %d: %s -- W: %d, P: %d, TPB: %d\n", 
			   i, device.name, device.warpSize, device.multiProcessorCount, device.maxThreadsPerBlock);
	}
	
	printf("END: Initializing Memory\n");
}

void vanity_run(config &vanity) {
	printf("Running on %d GPUs\n", vanity.gpuCount);
	
	// Allocate device memory for results
	int* dev_keys_found[MAX_NUM_GPUS] = {nullptr};
	int* dev_executions_this_gpu[MAX_NUM_GPUS] = {nullptr};
	
	for (int g = 0; g < vanity.gpuCount; ++g) {
		hipError_t err = hipSetDevice(g);
		if (err != hipSuccess) {
			fprintf(stderr, "Failed to set device %d: %s\n", g, hipGetErrorString(err));
			continue;
		}
		
		hipMalloc(&dev_keys_found[g], sizeof(int));
		hipMalloc(&dev_executions_this_gpu[g], sizeof(int));
		hipMemset(dev_keys_found[g], 0, sizeof(int));
		hipMemset(dev_executions_this_gpu[g], 0, sizeof(int));
		
		// Verify device is still set correctly
		int currentDevice;
		hipGetDevice(&currentDevice);
		printf("Initialized memory for GPU %d (current device: %d)\n", g, currentDevice);
	}

	unsigned long long int  executions_total = 0; 
	unsigned long long int  executions_this_iteration; 
	int  executions_this_gpu; 

	int  keys_found_total = 0;
	int  keys_found_this_iteration;

	for (int i = 0; i < MAX_ITERATIONS; ++i) {
		auto start  = std::chrono::high_resolution_clock::now();

                executions_this_iteration=0;

		// Reset counters at the start of each iteration
		for (int g = 0; g < vanity.gpuCount; ++g) {
			int zero = 0;
			hipMemcpy(dev_keys_found[g], &zero, sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(dev_executions_this_gpu[g], &zero, sizeof(int), hipMemcpyHostToDevice);
		}

		// Run on all GPUs
		for (int g = 0; g < vanity.gpuCount; ++g) {
			hipSetDevice(g);
			int blockSize = 0, minGridSize = 0, maxActiveBlocks = 0;
			hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, vanity_scan, 0, 0);
			hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocks, vanity_scan, blockSize, 0);

			int* dev_g;
	                hipMalloc((void**)&dev_g, sizeof(int));
                	hipMemcpy( dev_g, &g, sizeof(int), hipMemcpyHostToDevice ); 

			vanity_scan<<<maxActiveBlocks, blockSize>>>(vanity.states[g], dev_keys_found[g], dev_g, dev_executions_this_gpu[g]);
			hipFree(dev_g);
		}

		// Synchronize while we wait for kernels to complete. I do not
		// actually know if this will sync against all GPUs, it might
		// just sync with the last `i`, but they should all complete
		// roughly at the same time and worst case it will just stack
		// up kernels in the queue to run.
		hipDeviceSynchronize();
		auto finish = std::chrono::high_resolution_clock::now();

		for (int g = 0; g < vanity.gpuCount; ++g) {
                	hipMemcpy( &keys_found_this_iteration, dev_keys_found[g], sizeof(int), hipMemcpyDeviceToHost ); 
                	keys_found_total += keys_found_this_iteration; 
			//printf("GPU %d found %d keys\n",g,keys_found_this_iteration);

                	hipMemcpy(&executions_this_gpu, dev_executions_this_gpu[g], sizeof(int), hipMemcpyDeviceToHost);
					executions_this_iteration += executions_this_gpu * ATTEMPTS_PER_EXECUTION;
					executions_total += executions_this_gpu * ATTEMPTS_PER_EXECUTION;
                        //printf("GPU %d executions: %d\n",g,executions_this_gpu);
		}

		// Print out performance Summary
		std::chrono::duration<double> elapsed = finish - start;
		// printf("%s Iteration %d Attempts: %llu in %.2f at %.2f keys/sec - Total Attempts %llu - Keys Found %d\n",
		// 	getTimeStr().c_str(),
		// 	i+1,
		// 	executions_this_iteration,
		// 	elapsed.count(),
		// 	executions_this_iteration / elapsed.count(),
		// 	executions_total,
		// 	keys_found_total
		// );

                if ( keys_found_total >= STOP_AFTER_KEYS_FOUND ) {
                	printf("Enough keys found, Done! \n");
		        exit(0);	
		}	
	}

	// Cleanup device memory
	for (int g = 0; g < vanity.gpuCount; ++g) {
		hipFree(dev_keys_found[g]);
		hipFree(dev_executions_this_gpu[g]);
	}

	printf("Iterations complete, Done!\n");
}

/* -- CUDA Vanity Functions ------------------------------------------------- */

void __global__ vanity_init(unsigned long long int seed, hiprandState* state) {
	int id = threadIdx.x + (blockIdx.x * blockDim.x);  
	hiprand_init(seed, id, 0, &state[id]);
}

void __global__ vanity_scan(hiprandState* state, int* keys_found, int* gpu, int* exec_count) {
	int id = threadIdx.x + (blockIdx.x * blockDim.x);

    // Each thread only adds 1 to represent its ATTEMPTS_PER_EXECUTION
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *exec_count = gridDim.x * blockDim.x;
    }
    
    // SMITH - should really be passed in, but hey ho
    	int prefix_letter_counts[MAX_PATTERNS];
    	for (unsigned int n = 0; n < sizeof(prefixes) / sizeof(prefixes[0]); ++n) {
        	if ( MAX_PATTERNS == n ) {
            		printf("NEVER SPEAK TO ME OR MY SON AGAIN");
            		return;
        	}
        	int letter_count = 0;
        	for(; prefixes[n][letter_count]!=0; letter_count++);
        	prefix_letter_counts[n] = letter_count;
    	}

	// Local Kernel State
	ge_p3 A;
	hiprandState localState     = state[id];
	unsigned char seed[32]     = {0};
	unsigned char publick[32]  = {0};
	unsigned char privatek[64] = {0};
	char key[256]              = {0};
	//char pkey[256]             = {0};

	// Start from an Initial Random Seed (Slow)
	// NOTE: Insecure random number generator, do not use keys generator by
	// this program in live.
	// SMITH: localState should be entropy random now
	for (int i = 0; i < 32; ++i) {
		float random    = hiprand_uniform(&localState);
		uint8_t keybyte = (uint8_t)(random * 255);
		seed[i]         = keybyte;
	}

	// Generate Random Key Data
	sha512_context md;

	// I've unrolled all the MD5 calls and special cased them to 32 byte
	// inputs, which eliminates a lot of branching. This is a pretty poor
	// way to optimize GPU code though.
	//
	// A better approach would be to split this application into two
	// different kernels, one that is warp-efficient for SHA512 generation,
	// and another that is warp efficient for bignum division to more
	// efficiently scan for prefixes. Right now bs58enc cuts performance
	// from 16M keys on my machine per second to 4M.
	for (int attempts = 0; attempts < ATTEMPTS_PER_EXECUTION; ++attempts) {
		// sha512_init Inlined
		md.curlen   = 0;
		md.length   = 0;
		md.state[0] = UINT64_C(0x6a09e667f3bcc908);
		md.state[1] = UINT64_C(0xbb67ae8584caa73b);
		md.state[2] = UINT64_C(0x3c6ef372fe94f82b);
		md.state[3] = UINT64_C(0xa54ff53a5f1d36f1);
		md.state[4] = UINT64_C(0x510e527fade682d1);
		md.state[5] = UINT64_C(0x9b05688c2b3e6c1f);
		md.state[6] = UINT64_C(0x1f83d9abfb41bd6b);
		md.state[7] = UINT64_C(0x5be0cd19137e2179);

		// sha512_update inlined
		// 
		// All `if` statements from this function are eliminated if we
		// will only ever hash a 32 byte seed input. So inlining this
		// has a drastic speed improvement on GPUs.
		//
		// This means:
		//   * Normally we iterate for each 128 bytes of input, but we are always < 128. So no iteration.
		//   * We can eliminate a MIN(inlen, (128 - md.curlen)) comparison, specialize to 32, branch prediction improvement.
		//   * We can eliminate the in/inlen tracking as we will never subtract while under 128
		//   * As a result, the only thing update does is copy the bytes into the buffer.
		const unsigned char *in = seed;
		for (size_t i = 0; i < 32; i++) {
			md.buf[i + md.curlen] = in[i];
		}
		md.curlen += 32;


		// sha512_final inlined
		// 
		// As update was effectively elimiated, the only time we do
		// sha512_compress now is in the finalize function. We can also
		// optimize this:
		//
		// This means:
		//   * We don't need to care about the curlen > 112 check. Eliminating a branch.
		//   * We only need to run one round of sha512_compress, so we can inline it entirely as we don't need to unroll.
		md.length += md.curlen * UINT64_C(8);
		md.buf[md.curlen++] = (unsigned char)0x80;

		while (md.curlen < 120) {
			md.buf[md.curlen++] = (unsigned char)0;
		}

		STORE64H(md.length, md.buf+120);

		// Inline sha512_compress
		uint64_t S[8], W[80], t0, t1;
		int i;

		/* Copy state into S */
		for (i = 0; i < 8; i++) {
			S[i] = md.state[i];
		}

		/* Copy the state into 1024-bits into W[0..15] */
		for (i = 0; i < 16; i++) {
			LOAD64H(W[i], md.buf + (8*i));
		}

		/* Fill W[16..79] */
		for (i = 16; i < 80; i++) {
			W[i] = Gamma1(W[i - 2]) + W[i - 7] + Gamma0(W[i - 15]) + W[i - 16];
		}

		/* Compress */
		#define RND(a,b,c,d,e,f,g,h,i) \
		t0 = h + Sigma1(e) + Ch(e, f, g) + K[i] + W[i]; \
		t1 = Sigma0(a) + Maj(a, b, c);\
		d += t0; \
		h  = t0 + t1;

		for (i = 0; i < 80; i += 8) {
			RND(S[0],S[1],S[2],S[3],S[4],S[5],S[6],S[7],i+0);
			RND(S[7],S[0],S[1],S[2],S[3],S[4],S[5],S[6],i+1);
			RND(S[6],S[7],S[0],S[1],S[2],S[3],S[4],S[5],i+2);
			RND(S[5],S[6],S[7],S[0],S[1],S[2],S[3],S[4],i+3);
			RND(S[4],S[5],S[6],S[7],S[0],S[1],S[2],S[3],i+4);
			RND(S[3],S[4],S[5],S[6],S[7],S[0],S[1],S[2],i+5);
			RND(S[2],S[3],S[4],S[5],S[6],S[7],S[0],S[1],i+6);
			RND(S[1],S[2],S[3],S[4],S[5],S[6],S[7],S[0],i+7);
		}

		#undef RND

		/* Feedback */
		for (i = 0; i < 8; i++) {
			md.state[i] = md.state[i] + S[i];
		}

		// We can now output our finalized bytes into the output buffer.
		for (i = 0; i < 8; i++) {
			STORE64H(md.state[i], privatek+(8*i));
		}

		// Code Until here runs at 87_000_000H/s.

		// ed25519 Hash Clamping
		privatek[0]  &= 248;
		privatek[31] &= 63;
		privatek[31] |= 64;

		// ed25519 curve multiplication to extract a public key.
		ge_scalarmult_base(&A, privatek);
		ge_p3_tobytes(publick, &A);

		// Code Until here runs at 87_000_000H/s still!

		size_t keysize = 256;
		b58enc(key, &keysize, publick, 32);

		// Code Until here runs at 22_000_000H/s. b58enc badly needs optimization.

		// We don't have access to strncmp/strlen here, I don't know
		// what the efficient way of doing this on a GPU is, so I'll
		// start with a dumb loop. There seem to be implementations out
		// there of bignunm division done in parallel as a CUDA kernel
		// so it might make sense to write a new parallel kernel to do
		// this.

                for (int i = 0; i < sizeof(prefixes) / sizeof(prefixes[0]); ++i) {

                        for (int j = 0; j<prefix_letter_counts[i]; ++j) {

				// it doesn't match this prefix, no need to continue
				if ( !(prefixes[i][j] == '?') && !(prefixes[i][j] == key[j]) ) {
					break;
				}

                                // we got to the end of the prefix pattern, it matched!
                                if ( j == ( prefix_letter_counts[i] - 1) ) {
                                        atomicAdd(keys_found, 1);
                                        //size_t pkeysize = 256;
                                        //b58enc(pkey, &pkeysize, seed, 32);
                                       
				        // SMITH	
					// The 'key' variable is the public key in base58 'address' format
                                        // We display the seed in hex

					// Solana stores the keyfile as seed (first 32 bytes)
					// followed by public key (last 32 bytes)
					// as an array of decimal numbers in json format

                                        printf("GPU %d MATCH %s - ", *gpu, key);
                                        for(int n=0; n<sizeof(seed); n++) { 
						printf("%02x",(unsigned char)seed[n]); 
					}
					printf("\n");
					
                                        printf("[");
					for(int n=0; n<sizeof(seed); n++) { 
						printf("%d,",(unsigned char)seed[n]); 
					}
                                        for(int n=0; n<sizeof(publick); n++) {
					        if ( n+1==sizeof(publick) ) {	
							printf("%d",publick[n]);
						} else {
							printf("%d,",publick[n]);
						}
					}
                                        printf("]\n");

					/*
					printf("Public: ");
                                        for(int n=0; n<sizeof(publick); n++) { printf("%d ",publick[n]); }
					printf("\n");
					printf("Private: ");
                                        for(int n=0; n<sizeof(privatek); n++) { printf("%d ",privatek[n]); }
					printf("\n");
					printf("Seed: ");
                                        for(int n=0; n<sizeof(seed); n++) { printf("%d ",seed[n]); }
					printf("\n");
                                        */

                                        break;
				}

                        }
		}

		// Code Until here runs at 22_000_000H/s. So the above is fast enough.

		// Increment Seed.
		// NOTE: This is horrifically insecure. Please don't use these
		// keys on live. This increment is just so we don't have to
		// invoke the CUDA random number generator for each hash to
		// boost performance a little. Easy key generation, awful
		// security.
		for (int i = 0; i < 32; ++i) {
			if (seed[i] == 255) {
				seed[i]  = 0;
			} else {
				seed[i] += 1;
				break;
			}
		}
	}

	// Copy Random State so that future calls of this kernel/thread/block
	// don't repeat their sequences.
	state[id] = localState;
}

bool __device__ b58enc(
	char    *b58,
       	size_t  *b58sz,
       	uint8_t *data,
       	size_t  binsz
) {
	// Base58 Lookup Table
	const char b58digits_ordered[] = "123456789ABCDEFGHJKLMNPQRSTUVWXYZabcdefghijkmnopqrstuvwxyz";

	const uint8_t *bin = data;
	int carry;
	size_t i, j, high, zcount = 0;
	size_t size;
	
	while (zcount < binsz && !bin[zcount])
		++zcount;
	
	size = (binsz - zcount) * 138 / 100 + 1;
	uint8_t buf[256];
	memset(buf, 0, size);
	
	for (i = zcount, high = size - 1; i < binsz; ++i, high = j)
	{
		for (carry = bin[i], j = size - 1; (j > high) || carry; --j)
		{
			carry += 256 * buf[j];
			buf[j] = carry % 58;
			carry /= 58;
			if (!j) {
				// Otherwise j wraps to maxint which is > high
				break;
			}
		}
	}
	
	for (j = 0; j < size && !buf[j]; ++j);
	
	if (*b58sz <= zcount + size - j) {
		*b58sz = zcount + size - j + 1;
		return false;
	}
	
	if (zcount) memset(b58, '1', zcount);
	for (i = zcount; j < size; ++i, ++j) b58[i] = b58digits_ordered[buf[j]];

	b58[i] = '\0';
	*b58sz = i + 1;
	
	return true;
}
